/**
 * 
 *	This program computes the cost and the gradients of a sparse autoencoder
 *	neural network, to adjust its weights properly. It is a vectorized 
 *	implementation in CUDA C. It is a prototype and is only used to test
 *	the CUDA algorithm with a small set of artificial examples (artificial 
 *	dataset and weights).
 * 
 *
 *	compile it with nvcc -lcublas sparseAutoencoderCost.cu
 *
 *
 *	Author: Chistos Nikolaou
 *	Date: April 2014
 * 
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "computation_functions.h"

#define IND(i,j,ld) (((j)*(ld))+(i))

// Define functions
void SetInputVars(float *theta, float *data, int thetaLength, 
				  int numberOfExamples, int features);
void SetHostMatrices(int visibleSize, int hiddenSize, float *theta,
					 float *hostW1, float *hostW2, 
					 float *hostb1, float *hostb2);
void TestInputMatValues(int visibleSize, int hiddenSize, 
						float *W1, float *W2, float *b1, float *b2);
void SetDeviceMatrices(int visibleSize, int hiddenSize,
				float *hostW1, float *hostW2, float *hostb1, float *hostb2, 
				float *W1, float *W2, float *b1, float *b2);
void SetGradVec(int visibleSize, int hiddenSize, float *gradVec, 
				float *hostW1grad, float *hostW2grad, 
				float *hostb1grad, float *hostb2grad);

int main(void) {

	// set CUDA variables
	hipError_t cudaStat;
	hipblasStatus_t cublasStat;
	hipblasHandle_t handle;

	hipblasCreate(&handle);

	// These are inputs to the MATLAB code
	float *theta, *data;
	float lambda = 1;
	float sparsityParam = 0.1;
	float beta = 1;
	int visibleSize, hiddenSize;
	int numberOfExamples = 3;

	visibleSize = 10;
	hiddenSize = 4;

	// Define matrices
	float *W1, *W2, *b1, *b2;

	// allocate space for theta vector
	int thetaLength = (2*visibleSize*hiddenSize + hiddenSize + visibleSize);
	theta = (float *) malloc(thetaLength * sizeof(*theta));

	// allocate host memory for 
	data = (float *) malloc(numberOfExamples*visibleSize*sizeof(float));

	// print algorithm's information
	printf("Visible size = %d, ", visibleSize);
	printf("hidden size = %d, ", hiddenSize);
	printf("lambda = %f, ", lambda);
	printf("beta = %f, ", beta); 
	printf("sparsityParam = %f, ", sparsityParam);
	printf("thetaLength = %d\n", thetaLength);

	// set inputs for testing
	SetInputVars(theta, data, thetaLength, numberOfExamples, visibleSize);

	int i,j;

	printf("\n");
	printf("Matrix theta:\n");
	for(i = 0; i < thetaLength; i++) {
			printf("theta[%d] = %2.2f \n", i, theta[i]);
	}
	printf("\n");

	printf("DATA matrix\n");
	for(i = 0; i < visibleSize; i++) {
		for(j = 0; j < numberOfExamples; j++) {
			printf("dat[%d,%d]=%f ", i, j, data[IND(i,j,visibleSize)]);
		}
		printf("\n");
	}
	printf("\n");

	
	/* ----- Set host (weight) matrices from the theta vector ----- */
	float *hostW1, *hostW2, *hostb1, *hostb2;
	hostW1 = (float*) malloc(hiddenSize*visibleSize*sizeof(float));
	hostW2 = (float*) malloc(visibleSize*hiddenSize*sizeof(float));
	hostb1 = (float*) malloc(hiddenSize*sizeof(float));
	hostb2 = (float*) malloc(visibleSize*sizeof(float));

	SetHostMatrices(visibleSize, hiddenSize, 
					theta, hostW1, hostW2, hostb1, hostb2);

	
	/* ----- Matrix transfer to device ----- */

	// Memory space for W1 matrix
	cudaStat = hipMalloc((void**)&W1, visibleSize*hiddenSize*sizeof(float));
	if(cudaStat != hipSuccess) {
		printf("Unable to malloc memory on device for W1.\n");
		exit(1);
	}

	// Memory space for W2 matrix
 	cudaStat = hipMalloc((void**)&W2, visibleSize*hiddenSize*sizeof(float));
	if(cudaStat != hipSuccess) {
		printf("Unable to malloc memory on device for W2.\n");
		exit(1);
	}

	// Memory space for b1 matrix (vector)
	cudaStat = hipMalloc((void**)&b1, hiddenSize*sizeof(float));
	if(cudaStat != hipSuccess) {
		printf("Unable to malloc memory on device for b1.\n");
		exit(1);
	}

	// Memory space for b2 matrix (vector)
	cudaStat = hipMalloc((void**)&b2, visibleSize*sizeof(float));
	if(cudaStat != hipSuccess) {
		printf("Unable to malloc memory on device for b2.\n");
		exit(1);
	}

	SetDeviceMatrices(visibleSize, hiddenSize, 
					  hostW1, hostW2, hostb1, hostb2, W1, W2, b1, b2);


	/* ----- Define host matrices to test the values ----- */

	TestInputMatValues(visibleSize, hiddenSize, W1, W2, b1, b2);


	/* ----- Main program ----- */
	
	// Device memory allocation for the layer output matrices
	float *y, *x, *a1, *z2, *a2, *z3, *a3;

	cudaStat = hipMalloc((void**)&y, 
						  visibleSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&x, 
						  visibleSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&a1, 
						  visibleSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&z2, 
						  hiddenSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&a2,
						  hiddenSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&z3, 
						  visibleSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&a3, 
						  visibleSize*numberOfExamples*sizeof(float));


	/* ----- Forward Propagation ----- */

	float a = 1.0;
	float b = 1.0;

	// set input to be equal to data
	cublasStat = hipblasSetMatrix(visibleSize, numberOfExamples, sizeof(float),
								 data, visibleSize, x, visibleSize);
	if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to set x equal to input data.\n");
		exit(1);
	}

	// set target output y to be equal to inpute x.
	cublasStat = hipblasSetMatrix(visibleSize, numberOfExamples, sizeof(float),
								 data, visibleSize, y, visibleSize);
	if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to set y equal to input data (autoencoder).\n");
		exit(1);
	}

	// set z2 to repetition of b1 and compute 
	// z2 = W1*a1 + repmat(b1,1,numberOfExamples)
	SetRepMat(z2, hostb1, hiddenSize, numberOfExamples);

	// x equals a1
	cublasStat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, hiddenSize, 
							 numberOfExamples, visibleSize, &a, W1, hiddenSize,
							 x, visibleSize, &b, z2, hiddenSize); 
	
	if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to compute z2 = W1*a1 + z2 "); 
		printf("(=repmat(b1,1,numberOfExamples)).\n");
	}

	ComputeSigmoid(z2,a2,hiddenSize*numberOfExamples);

	// set z3 to repetition of b2 and compute 
	// z3 = W2*a2 + repmat(b2,1,numberOfExamples)
	SetRepMat(z3, hostb2, visibleSize, numberOfExamples);

	cublasStat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, visibleSize, 
							 numberOfExamples, hiddenSize, &a, W2, visibleSize, 
							 a2, hiddenSize, &b, z3, visibleSize);
			
	if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to compute z3 = W2*a2 + z3 ");
		printf("=repmap(b2,1,numberOfExamples)).\n");
	}

	ComputeSigmoid(z3,a3,visibleSize*numberOfExamples);


	/* --- Back Propagation ---*/

	// Parital Cost
	float *partCost, *delta3, *delta2;

	cudaStat = hipMalloc((void**)&partCost, 
						  visibleSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&delta2, 
						  hiddenSize*numberOfExamples*sizeof(float));
	cudaStat = hipMalloc((void**)&delta3, 
						  visibleSize*numberOfExamples*sizeof(float));

	ComputePartCost(handle,a3,y,partCost,visibleSize,numberOfExamples);

	// Delta
	dim3 d3Block(visibleSize*numberOfExamples);
	dim3 dimGrid(1,1);
	printf("Create block with %d threads: visibleSize*numberOfExamples\n", 
												visibleSize*numberOfExamples);

	CompDelta3<<<dimGrid,d3Block>>>(y,a3,delta3,visibleSize*numberOfExamples);

	CompDelta(handle,W2,delta3,a2,delta2,
			  hiddenSize,numberOfExamples,visibleSize);



	/* ----- Compute Error Gradients ----- */

	// Device memory allocation for the derivatives of weight matrices
	float *DW1, *DW2, *Db1, *Db2;

	cudaStat = hipMalloc((void**)&DW1, hiddenSize*visibleSize*sizeof(float));
	cudaStat = hipMalloc((void**)&Db1, hiddenSize*sizeof(float));
	cudaStat = hipMalloc((void**)&DW2, visibleSize*hiddenSize*sizeof(float));
	cudaStat = hipMalloc((void**)&Db2, visibleSize*sizeof(float));


	b = 0.0;

	// compute DW1 = delta2*a1'
	cublasStat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, hiddenSize, 
							 visibleSize, numberOfExamples,	&a, delta2, 
							 hiddenSize, x, visibleSize, &b, DW1, hiddenSize);

	// compute DW2 = delta3*a2'
	cublasStat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, visibleSize, 
							 hiddenSize, numberOfExamples, &a, delta3, 
							 visibleSize, a2, hiddenSize, &b, DW2, visibleSize);


	float *onesVec;

	// compute Db1 = sum(delta2,2)
	cudaStat = hipMalloc((void**)&onesVec, numberOfExamples*sizeof(float));

	dim3 onesBlock1(numberOfExamples,1);
	dim3 onesGrid1(1,1);
	printf("Create block with %d threads: numberOfExamples\n", 
												numberOfExamples);
	SetOnes<<<onesGrid1, onesBlock1>>>(onesVec,numberOfExamples);

	b = 0.0;

	cublasStat = hipblasSgemv(handle, HIPBLAS_OP_N, hiddenSize, 
							 numberOfExamples, &a, delta2, hiddenSize, 
							 onesVec, 1, &b, Db1, 1);

	// compute Db2 = sum(delta3,2) 

	b = 0.0;

	cublasStat = hipblasSgemv(handle, HIPBLAS_OP_N, visibleSize, 
							 numberOfExamples, &a, delta3, visibleSize, 
							 onesVec, 1, &b, Db2, 1);

	hipFree(onesVec);


	/* ----- Compute Cost ----- */

	float cost, *hostCost, *tempCost;

	cudaStat = hipMalloc((void**)&tempCost, sizeof(float));
	hostCost = (float*) malloc(sizeof(float));

	cudaStat = hipMalloc((void**)&onesVec, numberOfExamples*sizeof(float));

	dim3 onesBlock3(numberOfExamples,1);
	dim3 onesGrid3(1,1);
	SetOnes<<<onesGrid3,onesBlock3>>>(onesVec, numberOfExamples);

	b = 0.0;
	
	cublasStat = hipblasSgemv(handle, HIPBLAS_OP_T, numberOfExamples, 1,
							 &a, partCost, numberOfExamples, onesVec, 1, 
							 &b, tempCost, 1);

	cudaStat = hipMemcpy(hostCost, tempCost, sizeof(float), hipMemcpyDeviceToHost);

	cost = 1/(float)numberOfExamples * (*hostCost);



	/* ----- Compute gradients ----- */

	float *hostW1grad, *hostW2grad, *hostb1grad, *hostb2grad;

	hostW1grad = (float*) malloc(hiddenSize*visibleSize*sizeof(float));
	hostW2grad = (float*) malloc(visibleSize*hiddenSize*sizeof(float));
	hostb1grad = (float*) malloc(hiddenSize*sizeof(float));
	hostb2grad = (float*) malloc(visibleSize*sizeof(float));


	cublasStat = hipblasGetMatrix(hiddenSize, visibleSize, sizeof(float), 
								 DW1, hiddenSize, hostW1grad, hiddenSize);

	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR; Failed to copy DW1 device matrix to hostW1grad host matrix.\n");
		exit(1);
	}

	cublasStat = hipblasGetMatrix(visibleSize, hiddenSize, sizeof(float), 
								 DW2, visibleSize, hostW2grad, visibleSize);

	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR; Failed to copy DW1 device matrix to hostW1grad host matrix.\n");
		exit(1);
	}

	cublasStat = hipblasGetMatrix(hiddenSize, 1, sizeof(float), 
								 Db1, hiddenSize, hostb1grad, hiddenSize);

	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR; Failed to copy DW1 device matrix to hostW1grad host matrix.\n");
		exit(1);
	}

	cublasStat = hipblasGetMatrix(visibleSize, 1, sizeof(float), 
								 Db2, visibleSize, hostb2grad, visibleSize);

	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR; Failed to copy DW1 device matrix to hostW1grad host matrix.\n");
		exit(1);
	}


	// Set grafient final values
	CompWgrad(hostW1grad, hiddenSize, visibleSize, numberOfExamples);
	CompWgrad(hostW2grad, visibleSize, hiddenSize, numberOfExamples);
	Compbgrad(hostb1grad, hiddenSize, numberOfExamples);
	Compbgrad(hostb2grad, visibleSize, numberOfExamples);

	/* ----- Define the gradient vector (theta grad) ----- */


	float *gradVec;

	gradVec = (float*) malloc(thetaLength*sizeof(float));


	SetGradVec(visibleSize, hiddenSize, gradVec, 
			   hostW1grad, hostW2grad, hostb1grad, hostb2grad);



	/* ----- Print computed matrices for testing----- */
	printf("\nPrint matrix z2:\n");
	PrintReturnedMat(hiddenSize, numberOfExamples, z2);

	printf("\nPrint matrix a2:\n");
	PrintReturnedMat(hiddenSize, numberOfExamples, a2);

	printf("\nPrint matrix z3:\n");
	PrintReturnedMat(visibleSize, numberOfExamples, z3);

	printf("\nPrint matrix a3:\n");
	PrintReturnedMat(visibleSize, numberOfExamples, a3);

	printf("\nPrint matrix partCost:\n");
	PrintReturnedMat(numberOfExamples, 1, partCost);

	printf("\nPrint matrix delta3:\n");
	PrintReturnedMat(visibleSize, numberOfExamples, delta3);

	printf("\nPrint matrix delta2:\n");
	PrintReturnedMat(hiddenSize, numberOfExamples, delta2);

	printf("\nPrint matrix DW1:\n");
	PrintReturnedMat(hiddenSize, visibleSize, DW1);
	
	printf("\nPrint matrix DW2:\n");
	PrintReturnedMat(visibleSize, hiddenSize, DW2);

	printf("\nPrint matrix Db1:\n");
	PrintReturnedMat(hiddenSize, 1, Db1);

	printf("\nPrint matrix Db2:\n");
	PrintReturnedMat(visibleSize, 1, Db2);
	
	printf("\nPrint matrix tempCost:\n");
	PrintReturnedMat(1, 1, tempCost);

	printf("\nTotal cost is %f\n", cost);


	/* ----- Print grad vectort -----*/


	printf("\nTheta grad vector\n");
	printf("---------------------\n");
	for (i = 0; i < thetaLength; i++)
//		printf("i = %d : %f\n", i+1, gradVec[i]);



	/* ----- Free allocated memory ----- */
	hipblasDestroy(handle);
	
	hipFree(W1); hipFree(W2); hipFree(b1); hipFree(b2);
	hipFree(DW1); hipFree(DW2); hipFree(Db1); hipFree(Db2);
	hipFree(y); hipFree(x); hipFree(a1); hipFree(z2); hipFree(a2);
	hipFree(z3); hipFree(a3);

	hipFree(partCost); hipFree(delta2); hipFree(delta3);
}


void SetInputVars(float *theta, float *data, 
				  int thetaLength, int numberOfExamples, int features) {

	int i, j;

	for(i = 0; i < thetaLength; i++) {
		if(i < 100) 
			theta[i] = 0.01*i;
		else
			theta[i] = 0.99;
	}

	for(i = 0; i < features; i++) {
		for(j = 0; j < numberOfExamples; j++) {
			data[IND(i,j,features)] = 0.5;
			//printf("%d %d %d\n", i, j, IND(i,j,features));
		}
	}
}


void SetHostMatrices(int visibleSize, int hiddenSize, float *theta,
					 float *hostW1, float *hostW2, 
					 float *hostb1, float *hostb2) {

	int i,j;

	int offset = 0;

	printf("\nTo hostW1:\n");
	
	for(i = 0; i < hiddenSize; i++) {
		for(j = 0; j < visibleSize; j++) {
			hostW1[IND(i,j,hiddenSize)] = theta[i*visibleSize+j];
			printf("%d = %f \n", IND(i,j,hiddenSize), 
					theta[i*visibleSize+j]);
		}
	}
	
	offset += hiddenSize*visibleSize;

	
	printf("\nTo hostW2:\n");

	for(i = 0; i < visibleSize; i++) {
		for(j = 0; j < hiddenSize; j++) {
			hostW2[IND(i,j,visibleSize)] = theta[offset + i*hiddenSize+j];
			printf("%d = %f \n", IND(i,j,visibleSize), 
					theta[offset + i*hiddenSize+j]);
		}
	}
	
	offset += hiddenSize*visibleSize;

	
	printf("\nTo hostb1:\n");

	for(i = 0; i < hiddenSize; i++) {
		for(j = 0; j < 1; j++) {
			hostb1[IND(i,j,hiddenSize)] = theta[offset +  i + visibleSize*j];
			printf("%d = %f \n", IND(i,j,hiddenSize), 
					theta[offset + i + visibleSize*j]);
		}
	}
	
	offset += hiddenSize;

	
	printf("\nTo hostb2:\n");
	
	for(i = 0; i < visibleSize; i++) {
		for(j = 0; j < 1; j++) {
			hostb2[IND(i,j,visibleSize)] = theta[offset + i + hiddenSize*j];
			printf("%d = %f \n" , IND(i,j,hiddenSize), 
					theta[offset + i + hiddenSize*j]);
		}
	}

	offset += visibleSize;
	printf("\nOffset is %d\n", offset);
}


void TestInputMatValues(int visibleSize, int hiddenSize, 
						float *W1, float *W2, float *b1, float *b2) {
	
	hipblasStatus_t cublasStat;
	float *hostMat;
	int i,j;

	/* --- Print W1 matrix --- */

	// host memory space allocation fot the W1 matrix
	hostMat = (float *) malloc(visibleSize*hiddenSize*sizeof(float));

	// get elements for W1 matrix
	cublasStat = hipblasGetMatrix(hiddenSize, visibleSize, sizeof(float), 
								 W1, hiddenSize, hostMat, hiddenSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to get matrix W1.\n");
		exit(1);
	}

	// print W1 elements
	printf("Matrix W1:\n");
	for(i = 0; i < hiddenSize; i++) {
		for(j = 0; j < visibleSize; j++) {
			printf("W1[%d,%d] = %2.2f, ", i, j, hostMat[IND(i,j,hiddenSize)]);
		}
		printf("\n");
	}
	printf("\n");


	/* --- Print W2 matrix --- */
	
	// host memory space allocation for the W2 matrix
	hostMat = (float *) malloc(visibleSize*hiddenSize*sizeof(float));

	// get elements for W2 matrix
	cublasStat = hipblasGetMatrix(hiddenSize, visibleSize, sizeof(float), 
								 W2, hiddenSize, hostMat, hiddenSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to get matrix W2.\n");
		exit(1);
	}

	// print W2 elements
	printf("Matrix W2:\n");
	for(i = 0; i < visibleSize; i++) {
		for(j = 0; j < hiddenSize; j++) {
			printf("W2[%d,%d] = %2.2f, ", i, j, hostMat[i*hiddenSize+j]);
		}
		printf("\n");
	}
	printf("\n");


	/* --- Print b1 matrix --- */
	
	// host memory allocation foe the b1 matrix (vector)
	hostMat = (float *) malloc(hiddenSize*sizeof(float));

	// get elements fpr b2 matrix
	cublasStat = hipblasGetMatrix(hiddenSize, 1, sizeof(float), 
								 b1, hiddenSize, hostMat, hiddenSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to get matrix b1.\n");
		exit(1);
	}

	// printf b1 elements
	printf("Matrix b1:\n");
	for(i = 0; i < hiddenSize; i++) {
		printf("b1[%d] = %2.2f\n", i, hostMat[i]);
	}
	printf("\n");


	/* --- Print b2 matrix --- */

	// host memory allocation for the b2 matrix (vector)
	hostMat = (float *) malloc(visibleSize*sizeof(float));

	// get elements for b2 matrix
	cublasStat = hipblasGetMatrix(visibleSize, 1, sizeof(float), 
								 b2, visibleSize, hostMat, visibleSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to get matrix b2.\n");
		exit(1);
	}

	// print b2 elements
	printf("Matrix b2:\n");
	for(i = 0; i < visibleSize; i++) {
		printf("b2[%d] = %2.2f\n", i, hostMat[i]);
	}
	printf("\n");

	free(hostMat);
}


void SetDeviceMatrices(int visibleSize, int hiddenSize,
				float *hostW1, float *hostW2, float *hostb1, float *hostb2, 
				float *W1, float *W2, float *b1, float *b2) {

	hipblasStatus_t cublasStat;		

	// Set W1 device matrix
	cublasStat = hipblasSetMatrix(hiddenSize, visibleSize, sizeof(float), 
						     	 hostW1, hiddenSize, W1, hiddenSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to create matrix W1.\n");
		exit(1);
	}

	// Set W2 device matrix
	cublasStat = hipblasSetMatrix(visibleSize, hiddenSize, sizeof(float), 
								 hostW2, visibleSize, W2, visibleSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to create matrix W2.\n");
		exit(1);
	}
	
	// Set b1 device matrix (vector)
	cublasStat = hipblasSetMatrix(hiddenSize, 1, sizeof(float), 
								 hostb1, hiddenSize, b1, hiddenSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to create matrix b1.\n");
		exit(1);
	}

	// Set b2 device matrix (vector)
	cublasStat = hipblasSetMatrix(visibleSize, 1, sizeof(float), 
								 hostb2, visibleSize, b2, visibleSize);
	if(cublasStat != HIPBLAS_STATUS_SUCCESS) {
		printf("Unable to create matrix b2.\n");
		exit(1);
	}

}


void SetGradVec(int visibleSize, int hiddenSize, float *gradVec, 
				float *hostW1grad, float *hostW2grad, 
				float *hostb1grad, float *hostb2grad) {

	int i,j;

	int offset = 0;

	printf("\nFrom hostW1grad:\n");
	
	for(i = 0; i < hiddenSize; i++) {
		for(j = 0; j < visibleSize; j++) {
			gradVec[i*visibleSize+j] = hostW1grad[i*visibleSize+j]; 
			printf("position %d , place %f \n",	i*visibleSize+j,
				  	hostW1grad[i*visibleSize+j]);
		}
	}

	offset += hiddenSize*visibleSize;

	printf("\nFrom hostW2grad:\n");

	for(i = 0; i < visibleSize; i++) {
		for(j = 0; j < hiddenSize; j++) {
			gradVec[offset + i*hiddenSize + j] = hostW2grad[i*hiddenSize+j];
			printf("position %d , place %f \n", offset + i*hiddenSize + j, 
				   	hostW2grad[IND(i,j,visibleSize)]);
		}
	}

	offset += hiddenSize*visibleSize;

	
	printf("\nFrom hostb1grad:\n");
	
	for(i = 0; i < hiddenSize; i++) {
		for(j = 0; j < 1; j++) {
			gradVec[offset + i + visibleSize*j] = hostb1grad[i];
			printf("position %d , place %f \n",	offset + i + visibleSize*j, 
				   	hostb1grad[IND(i,j,hiddenSize)]);
		}
	}

	offset += hiddenSize;

	
	printf("\nFrom hostb2grad:\n");

	for(i = 0; i < visibleSize; i++) {
		for(j = 0; j < 1; j++) {
			gradVec[offset + i + hiddenSize*j] = hostb2grad[i];
			printf("position %d , place %f \n", offset + i + hiddenSize*j, 
				   	hostb2grad[IND(i,j,visibleSize)]);
		}
	}

	offset += visibleSize;
	printf("\nOffset is %d\n", offset);
}
